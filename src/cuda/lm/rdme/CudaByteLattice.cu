/*
 * University of Illinois Open Source License
 * Copyright 2008-2011 Luthey-Schulten Group,
 * All rights reserved.
 * 
 * Developed by: Luthey-Schulten Group
 * 			     University of Illinois at Urbana-Champaign
 * 			     http://www.scs.uiuc.edu/~schulten
 * 
 * Permission is hereby granted, free of charge, to any person obtaining a copy of
 * this software and associated documentation files (the Software), to deal with 
 * the Software without restriction, including without limitation the rights to 
 * use, copy, modify, merge, publish, distribute, sublicense, and/or sell copies 
 * of the Software, and to permit persons to whom the Software is furnished to 
 * do so, subject to the following conditions:
 * 
 * - Redistributions of source code must retain the above copyright notice, 
 * this list of conditions and the following disclaimers.
 * 
 * - Redistributions in binary form must reproduce the above copyright notice, 
 * this list of conditions and the following disclaimers in the documentation 
 * and/or other materials provided with the distribution.
 * 
 * - Neither the names of the Luthey-Schulten Group, University of Illinois at
 * Urbana-Champaign, nor the names of its contributors may be used to endorse or
 * promote products derived from this Software without specific prior written
 * permission.
 * 
 * THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR 
 * IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY, 
 * FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT.  IN NO EVENT SHALL 
 * THE CONTRIBUTORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR 
 * OTHER LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, 
 * ARISING FROM, OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR 
 * OTHER DEALINGS WITH THE SOFTWARE.
 *
 * Author(s): Elijah Roberts
 */

#include "lm/Types.h"
#include "lm/Exceptions.h"
#include "lm/rdme/ByteLattice.h"
#include "lm/rdme/CudaByteLattice.h"
#include "lm/rdme/Lattice.h"

namespace lm {
namespace rdme {

CudaByteLattice::CudaByteLattice(lattice_coord_t size, si_dist_t latticeSpacing, uint particlesPerSite)
throw(std::bad_alloc,InvalidArgException,Exception,CUDAException)
:ByteLattice(size,latticeSpacing,particlesPerSite),cudaParticlesCurrent(0),cudaParticlesSize(0),cudaSiteTypesSize(0),cudaSiteTypes(NULL),isGPUMemorySynched(false)
{
    // Initialize the pointers.
    cudaParticles[0] = NULL;
    cudaParticles[1] = NULL;

    // Make sure the lattice dimensions are divisible by 32.
    if (size.x%32 != 0 || size.y%32 != 0 || size.z%32 != 0) throw InvalidArgException("size","each dimension of a CUDA lattice must be divisible by 32");
    allocateCudaMemory();
}

CudaByteLattice::CudaByteLattice(lattice_size_t xSize, lattice_size_t ySize, lattice_size_t zSize, si_dist_t latticeSpacing, uint particlesPerSite)
throw(std::bad_alloc,InvalidArgException,Exception,CUDAException)
:ByteLattice(xSize,ySize,zSize,latticeSpacing,particlesPerSite),cudaParticlesCurrent(0),cudaParticlesSize(0),cudaSiteTypesSize(0),cudaSiteTypes(NULL),isGPUMemorySynched(false)
{
    // Initialize the pointers.
    cudaParticles[0] = NULL;
    cudaParticles[1] = NULL;

    // Make sure the lattice dimensions are divisible by 32.
    if (size.x%32 != 0 || size.y%32 != 0 || size.z%32 != 0) throw InvalidArgException("size","each dimension of a CUDA lattice must be divisible by 32");
    allocateCudaMemory();
}

CudaByteLattice::~CudaByteLattice()
throw(std::bad_alloc)
{
    deallocateCudaMemory();
}

void CudaByteLattice::allocateCudaMemory()
throw(CUDAException)
{
    // Allocate memory on the CUDA device.
    cudaParticlesSize=numberSites*wordsPerSite*sizeof(uint32_t);
    CUDA_EXCEPTION_CHECK(hipMalloc(&cudaParticles[0], cudaParticlesSize)); //TODO: track memory usage.
    CUDA_EXCEPTION_CHECK(hipMalloc(&cudaParticles[1], cudaParticlesSize)); //TODO: track memory usage.
    cudaSiteTypesSize=numberSites*sizeof(uint8_t);
    CUDA_EXCEPTION_CHECK(hipMalloc(&cudaSiteTypes, cudaSiteTypesSize)); //TODO: track memory usage.
}

void CudaByteLattice::deallocateCudaMemory()
throw(CUDAException)
{
    // If we have any allocated device memory, free it.
    if (cudaParticles[0] != NULL)
    {
        CUDA_EXCEPTION_CHECK(hipFree(cudaParticles[0])); //TODO: track memory usage.
        cudaParticles[0] = NULL;
    }
    if (cudaParticles[1] != NULL)
    {
        CUDA_EXCEPTION_CHECK(hipFree(cudaParticles[1])); //TODO: track memory usage.
        cudaParticles[1] = NULL;
    }
    cudaParticlesSize = 0;
    if (cudaSiteTypes != NULL)
    {
        CUDA_EXCEPTION_CHECK(hipFree(cudaSiteTypes)); //TODO: track memory usage.
        cudaSiteTypes = NULL;
        cudaSiteTypesSize = 0;
    }
}

void CudaByteLattice::copyToGPU()
throw(CUDAException)
{
	if (!isGPUMemorySynched)
	{
		CUDA_EXCEPTION_CHECK(hipMemcpy(cudaParticles[cudaParticlesCurrent], particles, cudaParticlesSize, hipMemcpyHostToDevice));
        CUDA_EXCEPTION_CHECK(hipMemcpy(cudaSiteTypes, siteTypes, cudaSiteTypesSize, hipMemcpyHostToDevice));
		isGPUMemorySynched = true;
	}
}

void CudaByteLattice::copyFromGPU()
throw(CUDAException)
{
	CUDA_EXCEPTION_CHECK(hipMemcpy(particles, cudaParticles[cudaParticlesCurrent], cudaParticlesSize, hipMemcpyDeviceToHost));
    CUDA_EXCEPTION_CHECK(hipMemcpy(siteTypes, cudaSiteTypes, cudaSiteTypesSize, hipMemcpyDeviceToHost));
	isGPUMemorySynched = true;
}

void * CudaByteLattice::getGPUMemorySrc()
{
    return cudaParticles[cudaParticlesCurrent];
}

void * CudaByteLattice::getGPUMemoryDest()
{
    return cudaParticles[cudaParticlesCurrent==0?1:0];
}

void CudaByteLattice::swapSrcDest()
{
    cudaParticlesCurrent = cudaParticlesCurrent==0?1:0;
}

void * CudaByteLattice::getGPUMemorySiteTypes()
{
    return cudaSiteTypes;
}

void CudaByteLattice::setSiteType(lattice_size_t x, lattice_size_t y, lattice_size_t z, site_t site) throw(InvalidSiteException)
{
    ByteLattice::setSiteType(x,y,z,site);
    isGPUMemorySynched = false;
}

void CudaByteLattice::addParticle(lattice_size_t x, lattice_size_t y, lattice_size_t z, particle_t particle) throw(InvalidSiteException,InvalidParticleException)
{
    ByteLattice::addParticle(x,y,z,particle);
	isGPUMemorySynched = false;
}

void CudaByteLattice::removeParticles(lattice_size_t x,lattice_size_t y,lattice_size_t z) throw(InvalidSiteException)
{
    ByteLattice::removeParticles(x,y,z);
    isGPUMemorySynched = false;
}

void CudaByteLattice::setSiteType(lattice_size_t index, site_t site) throw(InvalidSiteException)
{
    ByteLattice::setSiteType(index,site);
    isGPUMemorySynched = false;
}

void CudaByteLattice::addParticle(lattice_size_t index, particle_t particle) throw(InvalidSiteException,InvalidParticleException)
{
    ByteLattice::addParticle(index,particle);
	isGPUMemorySynched = false;
}

void CudaByteLattice::removeParticles(lattice_size_t index) throw(InvalidSiteException)
{
    ByteLattice::removeParticles(index);
    isGPUMemorySynched = false;
}

void CudaByteLattice::removeAllParticles()
{
    ByteLattice::removeAllParticles();
	isGPUMemorySynched = false;
}

void CudaByteLattice::deserializeParticlesFrom(const void* srcBuffer, size_t bufferSize, SerializationDataOrder dataOrdering, bool inflate)
{
    ByteLattice::deserializeParticlesFrom(srcBuffer, bufferSize, dataOrdering, inflate);
    isGPUMemorySynched = false;
}

}
}
