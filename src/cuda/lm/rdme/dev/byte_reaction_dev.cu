#include "hip/hip_runtime.h"
/*
 * University of Illinois Open Source License
 * Copyright 2008-2011 Luthey-Schulten Group,
 * All rights reserved.
 *
 * Developed by: Luthey-Schulten Group
 *               University of Illinois at Urbana-Champaign
 *               http://www.scs.uiuc.edu/~schulten
 *
 * Permission is hereby granted, free of charge, to any person obtaining a copy of
 * this software and associated documentation files (the Software), to deal with
 * the Software without restriction, including without limitation the rights to
 * use, copy, modify, merge, publish, distribute, sublicense, and/or sell copies
 * of the Software, and to permit persons to whom the Software is furnished to
 * do so, subject to the following conditions:
 *
 * - Redistributions of source code must retain the above copyright notice,
 * this list of conditions and the following disclaimers.
 *
 * - Redistributions in binary form must reproduce the above copyright notice,
 * this list of conditions and the following disclaimers in the documentation
 * and/or other materials provided with the distribution.
 *
 * - Neither the names of the Luthey-Schulten Group, University of Illinois at
 * Urbana-Champaign, nor the names of its contributors may be used to endorse or
 * promote products derived from this Software without specific prior written
 * permission.
 *
 * THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
 * IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
 * FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT.  IN NO EVENT SHALL
 * THE CONTRIBUTORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR
 * OTHER LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE,
 * ARISING FROM, OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR
 * OTHER DEALINGS WITH THE SOFTWARE.
 *
 * Author(s): Elijah Roberts
 */

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#if !defined MPD_WORDS_PER_SITE
#error "Must define the number of words per site."
#endif

#define MPD_PARTICLES_PER_SITE MPD_WORDS_PER_SITE*4

#if !defined TUNE_MPD_MAX_PARTICLE_OVERFLOWS
#error "Must define the maximum size of overflow particle list."
#endif

#define MPD_FIRST_ORDER_REACTION        1
#define MPD_SECOND_ORDER_REACTION       2
#define MPD_SECOND_ORDER_SELF_REACTION  3

#define MPD_MAX_REACTION_TABLE_ENTRIES	32
//TODO: check for performance hit/advantage if using byte arrays for constants.
__device__ __constant__ unsigned int numberReactionsC;
__device__ __constant__ unsigned int reactionOrdersC[MPD_MAX_REACTION_TABLE_ENTRIES];
__device__ __constant__ unsigned int reactionSitesC[MPD_MAX_REACTION_TABLE_ENTRIES];
__device__ __constant__ unsigned int D1C[MPD_MAX_REACTION_TABLE_ENTRIES];
__device__ __constant__ unsigned int D2C[MPD_MAX_REACTION_TABLE_ENTRIES];
__device__ __constant__ float reactionRatesC[MPD_MAX_REACTION_TABLE_ENTRIES];

/**
 * The stoichiometric matrix: numberSpecies x numberReactions
 */
#define MPD_MAX_S_MATRIX_ENTRIES    	512
__device__ __constant__ int8_t SC[MPD_MAX_S_MATRIX_ENTRIES];

/**
 * The reaction location matrix: numberReaction x numberSiteTypes
 */
#define MPD_MAX_RL_MATRIX_ENTRIES    	320
__device__ __constant__ uint8_t RLC[MPD_MAX_RL_MATRIX_ENTRIES];

inline __device__ float calculateReactionPropensity(const uint8_t siteType, const uint8_t * __restrict__ particles, const unsigned int reactionIndex)
{
    // Make sure that the reaction is valid for this site type.
    if (!RLC[reactionIndex*numberSiteTypesC+siteType]) return 0.0f;

    // Get the number of each reactant.
    float numberParticles1 = 0.0f;
    float numberParticles2 = 0.0f;
    for (int i=0; i<MPD_PARTICLES_PER_SITE; i++)
    {
        if (particles[i] > 0)
        {
            numberParticles1 += (particles[i] == D1C[reactionIndex])?(1.0f):(0.0f);
            numberParticles2 += (particles[i] == D2C[reactionIndex])?(1.0f):(0.0f);
        }
    }

    // Calculate the propensity according to the reaction order.
    if (reactionOrdersC[reactionIndex] == MPD_FIRST_ORDER_REACTION)
        return reactionRatesC[reactionIndex]*numberParticles1;
    else if (reactionOrdersC[reactionIndex] == MPD_SECOND_ORDER_REACTION)
        return reactionRatesC[reactionIndex]*numberParticles1*numberParticles2;
    else if (reactionOrdersC[reactionIndex] == MPD_SECOND_ORDER_SELF_REACTION)
        return reactionRatesC[reactionIndex]*numberParticles1*(numberParticles1-1.0f);
    return 0.0f;
}

inline __device__ float calculateReactionProbability(const float rate)
{
    #ifdef CUDA_DOUBLE_PRECISION
    return (float)(1.0-exp(-(double)rate));
    #else
    return (rate > 2e-4f)?(1.0f-__expf(-rate)):(rate);
    #endif
}

inline __device__ unsigned int checkForReaction(const unsigned int latticeIndex, const float reactionProbability, const unsigned long long timestepHash)
{
    return reactionProbability > 0.0f && getRandomHashFloat(latticeIndex, 1, 0, timestepHash) <= reactionProbability;
}

inline __device__ unsigned int determineReactionIndex(const uint8_t siteType, const uint8_t * __restrict__ particles, const unsigned int latticeIndex, const float totalReactionPropensity, const unsigned long long timestepHash)
{
    float randomPropensity = getRandomHashFloat(latticeIndex, 1, 1, timestepHash)*totalReactionPropensity;
    unsigned int reactionIndex = 0;
    for (int i=0; i<numberReactionsC; i++)
    {
        float propensity = calculateReactionPropensity(siteType, particles, i);
        if (propensity > 0.0f)
        {
            if (randomPropensity > 0.0f)
                reactionIndex = i;
            randomPropensity -= propensity;
        }

    }
    return reactionIndex;
}

__device__ void evaluateReaction(const unsigned int latticeIndex, const uint8_t siteType, uint8_t * __restrict__ particles, const unsigned int reactionIndex, unsigned int * siteOverflowList)
{
	// Copy the S matrix entries for this reaction.
    #if __CUDA_ARCH__ >= 200
	int8_t * S = (int8_t *)malloc(numberSpeciesC);
    #else
    int8_t S[256];
    #endif
	for (uint i=0, index=reactionIndex; i<numberSpeciesC; i++, index+=numberReactionsC)
		S[i] = SC[index];

    // Build the new site, copying particles that didn't react and removing those that did.
    int nextParticle=0;
    for (uint i=0; i<MPD_PARTICLES_PER_SITE; i++)
    {
    	uint8_t particle = particles[i];
        if (particle > 0)
        {
        	// If this particle was unaffected, copy it.
        	if (S[particle-1] >= 0)
        	{
        		particles[nextParticle++] = particle;
        	}

            // Otherwise, don't copy the particle and mark that we destroyed it.
        	else
        	{
        		S[particle-1]++;
        	}
        }
    }

    // Go through the S matrix and add in any new particles that were created.
    for (uint i=0; i<numberSpeciesC; i++)
    {
		for (uint j=0; j<S[i]; j++)
		{
			// If the particle will fit into the site, add it.
			if (nextParticle < MPD_PARTICLES_PER_SITE)
			{
				particles[nextParticle++] = i+1;
			}

			// Otherwise add it to the exception list.
			else
			{
				int exceptionIndex = atomicAdd(siteOverflowList, 1);
				if (exceptionIndex < TUNE_MPD_MAX_PARTICLE_OVERFLOWS)
				{
					siteOverflowList[(exceptionIndex*2)+1]=latticeIndex;
					siteOverflowList[(exceptionIndex*2)+2]=i+1;
				}
			}

		}
    }

    // Clear any remaining particles in the site.
    while (nextParticle < MPD_PARTICLES_PER_SITE)
    	particles[nextParticle++] = 0;

    // Free any allocated memory.
    #if __CUDA_ARCH__ >= 200
    free(S);
    #endif
}
