#include "hip/hip_runtime.h"
/*
 * University of Illinois Open Source License
 * Copyright 2010 Luthey-Schulten Group,
 * All rights reserved.
 *
 * Developed by: Luthey-Schulten Group
 * 			     University of Illinois at Urbana-Champaign
 * 			     http://www.scs.uiuc.edu/~schulten
 *
 * Permission is hereby granted, free of charge, to any person obtaining a copy of
 * this software and associated documentation files (the Software), to deal with
 * the Software without restriction, including without limitation the rights to
 * use, copy, modify, merge, publish, distribute, sublicense, and/or sell copies
 * of the Software, and to permit persons to whom the Software is furnished to
 * do so, subject to the following conditions:
 *
 * - Redistributions of source code must retain the above copyright notice,
 * this list of conditions and the following disclaimers.
 *
 * - Redistributions in binary form must reproduce the above copyright notice,
 * this list of conditions and the following disclaimers in the documentation
 * and/or other materials provided with the distribution.
 *
 * - Neither the names of the Luthey-Schulten Group, University of Illinois at
 * Urbana-Champaign, nor the names of its contributors may be used to endorse or
 * promote products derived from this Software without specific prior written
 * permission.
 *
 * THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
 * IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
 * FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT.  IN NO EVENT SHALL
 * THE CONTRIBUTORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR
 * OTHER LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE,
 * ARISING FROM, OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR
 * OTHER DEALINGS WITH THE SOFTWARE.
 *
 * Author(s): Elijah Roberts
 */

#include "lm/Cuda.h"

#define LS_WORDS_PER_SITE               2
#define LS_APRON_SIZE                   2
#define LS_X_BLOCK_MAX_X_SIZE           128
#define LS_Y_BLOCK_X_SIZE               16
#define LS_Y_BLOCK_Y_SIZE               8
#define LS_Z_BLOCK_X_SIZE               16
#define LS_Z_BLOCK_Z_SIZE               8
#define LS_BOUNDARY_VALUE               0xFFEEDDCC

#include "lm/rdme/dev/lattice_sim_1d_dev.cu"

__global__ void cu_CopyXWindowSites_kernel(const unsigned int* inLattice, unsigned int* outLattice, const unsigned int gridXSize, const unsigned int latticeXSize, const unsigned int latticeYSize, const unsigned int latticeXYZSize);
__global__ void cu_CopyXWindowAprons_kernel(const unsigned int* inLattice, unsigned int* outLattice, const unsigned int gridXSize, const unsigned int latticeXSize, const unsigned int latticeYSize, const unsigned int latticeXYZSize);

void cu_CopyXWindowSites(unsigned int * host_inLattice, unsigned int * host_outLattice, unsigned int latticeXSize, unsigned int latticeYSize, unsigned int latticeZSize)
{
    void* inLattice;
    void* outLattice;
    CUDA_EXCEPTION_CHECK(hipMalloc(&inLattice, latticeXSize*latticeYSize*latticeZSize*LS_WORDS_PER_SITE*sizeof(unsigned int)));
    CUDA_EXCEPTION_CHECK(hipMalloc(&outLattice, latticeXSize*latticeYSize*latticeZSize*LS_WORDS_PER_SITE*sizeof(unsigned int)));
    CUDA_EXCEPTION_CHECK(hipMemcpy(inLattice, host_inLattice, latticeXSize*latticeYSize*latticeZSize*LS_WORDS_PER_SITE*sizeof(unsigned int), hipMemcpyHostToDevice));
    CUDA_EXCEPTION_CHECK(hipMemset(outLattice, 0xFF, latticeXSize*latticeYSize*latticeZSize*LS_WORDS_PER_SITE*sizeof(unsigned int)));

    unsigned int gridXSize;
    dim3 gridSize, threadBlockSize;
    calculateXLaunchParameters(&gridXSize, &gridSize, &threadBlockSize, LS_X_BLOCK_MAX_X_SIZE, latticeXSize, latticeYSize, latticeZSize);
    CUDA_EXCEPTION_EXECUTE((cu_CopyXWindowSites_kernel<<<gridSize,threadBlockSize>>>((unsigned int*)inLattice, (unsigned int*)outLattice, gridXSize, latticeXSize, latticeYSize, latticeXSize*latticeYSize*latticeZSize)));

    CUDA_EXCEPTION_CHECK(hipStreamSynchronize(0));
    CUDA_EXCEPTION_CHECK(hipMemcpy(host_outLattice, outLattice, latticeXSize*latticeYSize*latticeZSize*LS_WORDS_PER_SITE*sizeof(unsigned int), hipMemcpyDeviceToHost));

    CUDA_EXCEPTION_CHECK(hipFree(outLattice));
    CUDA_EXCEPTION_CHECK(hipFree(inLattice));
}

__global__ void cu_CopyXWindowSites_kernel(const unsigned int* inLattice, unsigned int* outLattice, const unsigned int gridXSize, const unsigned int latticeXSize, const unsigned int latticeYSize, const unsigned int latticeXYZSize)
{
    __shared__ unsigned int bx, by, bz;
    calculateBlockPosition(&bx, &by, &bz, gridXSize);

    // Figure out the offset of this thread in the lattice and the lattice segment.
    unsigned int latticeXIndex = (bx*blockDim.x) + threadIdx.x;
    unsigned int latticeIndex = (bz*latticeXSize*latticeYSize) + (by*latticeXSize) + latticeXIndex;
    unsigned int windowIndex = threadIdx.x+LS_APRON_SIZE;

    ///////////////////////////////////////////
    // Load the lattice into shared memory. //
    ///////////////////////////////////////////

    // Shared memory to store the lattice segment. Each lattice site has four particles, eight bits for each particle.
    __shared__ unsigned int window[LS_X_WINDOW_SIZE*LS_WORDS_PER_SITE];

    // Copy the x window from device memory into shared memory.
    copyXWindowFromLattice(bx, inLattice, window, latticeIndex, latticeXIndex, latticeXSize, latticeXYZSize, windowIndex);

    // Copy the x window from shared memory to device memory.
    copyXWindowToLattice(outLattice, window, latticeIndex, latticeXIndex, latticeXSize, latticeXYZSize, windowIndex);
}

void cu_CopyXWindowAprons(unsigned int * host_inLattice, unsigned int * host_outLattice, unsigned int latticeXSize, unsigned int latticeYSize, unsigned int latticeZSize)
{
    void* inLattice;
    void* outLattice;
    CUDA_EXCEPTION_CHECK(hipMalloc(&inLattice, latticeXSize*latticeYSize*latticeZSize*LS_WORDS_PER_SITE*sizeof(unsigned int)));
    CUDA_EXCEPTION_CHECK(hipMalloc(&outLattice, latticeXSize*latticeYSize*latticeZSize*LS_WORDS_PER_SITE*sizeof(unsigned int)));
    CUDA_EXCEPTION_CHECK(hipMemcpy(inLattice, host_inLattice, latticeXSize*latticeYSize*latticeZSize*LS_WORDS_PER_SITE*sizeof(unsigned int), hipMemcpyHostToDevice));
    CUDA_EXCEPTION_CHECK(hipMemset(outLattice, 0xFF, latticeXSize*latticeYSize*latticeZSize*LS_WORDS_PER_SITE*sizeof(unsigned int)));

    unsigned int gridXSize;
    dim3 gridSize, threadBlockSize;
    calculateXLaunchParameters(&gridXSize, &gridSize, &threadBlockSize, LS_X_BLOCK_MAX_X_SIZE, latticeXSize, latticeYSize, latticeZSize);
    CUDA_EXCEPTION_EXECUTE((cu_CopyXWindowAprons_kernel<<<gridSize,threadBlockSize>>>((unsigned int*)inLattice, (unsigned int*)outLattice, gridXSize, latticeXSize, latticeYSize, latticeXSize*latticeYSize*latticeZSize)));

    CUDA_EXCEPTION_CHECK(hipStreamSynchronize(0));
    CUDA_EXCEPTION_CHECK(hipMemcpy(host_outLattice, outLattice, latticeXSize*latticeYSize*latticeZSize*LS_WORDS_PER_SITE*sizeof(unsigned int), hipMemcpyDeviceToHost));

    CUDA_EXCEPTION_CHECK(hipFree(outLattice));
    CUDA_EXCEPTION_CHECK(hipFree(inLattice));
}

__global__ void cu_CopyXWindowAprons_kernel(const unsigned int* inLattice, unsigned int* outLattice, const unsigned int gridXSize, const unsigned int latticeXSize, const unsigned int latticeYSize, const unsigned int latticeXYZSize)
{
    __shared__ unsigned int bx, by, bz;
    calculateBlockPosition(&bx, &by, &bz, gridXSize);

    // Figure out the offset of this thread in the lattice and the lattice segment.
    unsigned int latticeXIndex = (bx*blockDim.x) + threadIdx.x;
    unsigned int latticeIndex = (bz*latticeXSize*latticeYSize) + (by*latticeXSize) + latticeXIndex;
    unsigned int windowIndex = threadIdx.x+LS_APRON_SIZE;

    ///////////////////////////////////////////
    // Load the lattice into shared memory. //
    ///////////////////////////////////////////

    // Shared memory to store the lattice segment. Each lattice site has four particles, eight bits for each particle.
    __shared__ unsigned int window[LS_X_WINDOW_SIZE*LS_WORDS_PER_SITE];

    // Copy the x window from device memory into shared memory.
    copyXWindowFromLattice(bx, inLattice, window, latticeIndex, latticeXIndex, latticeXSize, latticeXYZSize, windowIndex);

    __syncthreads();

    if (latticeXIndex < latticeXSize)
    {
        outLattice[latticeIndex] = 0;
        outLattice[latticeIndex+latticeXYZSize] = 0;

        // If this is the first part of the block, load the leading apron.
        if (windowIndex < 2*LS_APRON_SIZE)
        {
            outLattice[latticeIndex] = window[windowIndex-LS_APRON_SIZE];
            outLattice[latticeIndex+latticeXYZSize] = window[windowIndex-LS_APRON_SIZE+LS_X_WINDOW_SIZE];
        }

        // If this is the last part of the block, load the trailing apron.
        if (windowIndex >= blockDim.x)
        {
            outLattice[latticeIndex] = window[windowIndex+LS_APRON_SIZE];
            outLattice[latticeIndex+latticeXYZSize] = window[windowIndex+LS_APRON_SIZE+LS_X_WINDOW_SIZE];
        }
    }
}
