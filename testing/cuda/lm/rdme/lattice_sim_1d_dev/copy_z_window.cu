#include "hip/hip_runtime.h"
/*
 * University of Illinois Open Source License
 * Copyright 2010 Luthey-Schulten Group,
 * All rights reserved.
 *
 * Developed by: Luthey-Schulten Group
 * 			     University of Illinois at Urbana-Champaign
 * 			     http://www.scs.uiuc.edu/~schulten
 *
 * Permission is hereby granted, free of charge, to any person obtaining a copy of
 * this software and associated documentation files (the Software), to deal with
 * the Software without restriction, including without limitation the rights to
 * use, copy, modify, merge, publish, distribute, sublicense, and/or sell copies
 * of the Software, and to permit persons to whom the Software is furnished to
 * do so, subject to the following conditions:
 *
 * - Redistributions of source code must retain the above copyright notice,
 * this list of conditions and the following disclaimers.
 *
 * - Redistributions in binary form must reproduce the above copyright notice,
 * this list of conditions and the following disclaimers in the documentation
 * and/or other materials provided with the distribution.
 *
 * - Neither the names of the Luthey-Schulten Group, University of Illinois at
 * Urbana-Champaign, nor the names of its contributors may be used to endorse or
 * promote products derived from this Software without specific prior written
 * permission.
 *
 * THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
 * IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
 * FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT.  IN NO EVENT SHALL
 * THE CONTRIBUTORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR
 * OTHER LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE,
 * ARISING FROM, OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR
 * OTHER DEALINGS WITH THE SOFTWARE.
 *
 * Author(s): Elijah Roberts
 */

#include "lm/Cuda.h"

#define LS_WORDS_PER_SITE               2
#define LS_APRON_SIZE            2
#define LS_X_BLOCK_MAX_X_SIZE           128
#define LS_Y_BLOCK_X_SIZE               16
#define LS_Y_BLOCK_Y_SIZE               8
#define LS_Z_BLOCK_X_SIZE               16
#define LS_Z_BLOCK_Z_SIZE               8
#define LS_BOUNDARY_VALUE               0xFFEEDDCC

#include "lm/rdme/dev/lattice_sim_1d_dev.cu"

__global__ void cu_CopyZWindowSites_kernel(const unsigned int* inLattice, unsigned int* outLattice, const unsigned int gridXSize, const unsigned int latticeXSize, const unsigned int latticeYSize, const unsigned int latticeZSize);
__global__ void cu_CopyZWindowAprons_kernel(const unsigned int* inLattice, unsigned int* outLattice, const unsigned int gridXSize, const unsigned int latticeXSize, const unsigned int latticeYSize, const unsigned int latticeZSize);

void cu_CopyZWindowSites(unsigned int * host_inLattice, unsigned int * host_outLattice, unsigned int latticeXSize, unsigned int latticeYSize, unsigned int latticeZSize)
{
    void* inLattice;
    void* outLattice;
    CUDA_EXCEPTION_CHECK(hipMalloc(&inLattice, latticeXSize*latticeYSize*latticeZSize*LS_WORDS_PER_SITE*sizeof(unsigned int)));
    CUDA_EXCEPTION_CHECK(hipMalloc(&outLattice, latticeXSize*latticeYSize*latticeZSize*LS_WORDS_PER_SITE*sizeof(unsigned int)));
    CUDA_EXCEPTION_CHECK(hipMemcpy(inLattice, host_inLattice, latticeXSize*latticeYSize*latticeZSize*LS_WORDS_PER_SITE*sizeof(unsigned int), hipMemcpyHostToDevice));
    CUDA_EXCEPTION_CHECK(hipMemset(outLattice, 0xFF, latticeXSize*latticeYSize*latticeZSize*LS_WORDS_PER_SITE*sizeof(unsigned int)));

    unsigned int gridXSize = latticeXSize/LS_Z_BLOCK_X_SIZE;
    unsigned int gridYSize = latticeYSize;
    unsigned int gridZSize = latticeZSize/LS_Z_BLOCK_Z_SIZE;
    dim3 gridSize(gridXSize*gridYSize, gridZSize);
    dim3 threadBlockSize(LS_Z_BLOCK_X_SIZE, 1, LS_Z_BLOCK_Z_SIZE);
    CUDA_EXCEPTION_EXECUTE((cu_CopyZWindowSites_kernel<<<gridSize,threadBlockSize>>>((unsigned int*)inLattice, (unsigned int*)outLattice, gridXSize, latticeXSize, latticeYSize, latticeZSize)));

    CUDA_EXCEPTION_CHECK(hipStreamSynchronize(0));
    CUDA_EXCEPTION_CHECK(hipMemcpy(host_outLattice, outLattice, latticeXSize*latticeYSize*latticeZSize*LS_WORDS_PER_SITE*sizeof(unsigned int), hipMemcpyDeviceToHost));

    CUDA_EXCEPTION_CHECK(hipFree(outLattice));
    CUDA_EXCEPTION_CHECK(hipFree(inLattice));
}

__global__ void cu_CopyZWindowSites_kernel(const unsigned int* inLattice, unsigned int* outLattice, const unsigned int gridXSize, const unsigned int latticeXSize, const unsigned int latticeYSize, const unsigned int latticeZSize)
{
    unsigned int latticeXYSize = latticeXSize*latticeYSize;
    unsigned int latticeXYZSize = latticeXSize*latticeYSize*latticeZSize;

    __shared__ unsigned int bx, by, bz;
    calculateBlockPosition(&bx, &by, &bz, gridXSize);

    // Figure out the offset of this thread in the lattice and the lattice segment.
    unsigned int latticeZIndex = (bz*blockDim.z) + threadIdx.z;
    unsigned int latticeIndex = (latticeZIndex*latticeXYSize) + (by*latticeXSize) + (bx*blockDim.x) + threadIdx.x;
    unsigned int windowZIndex = threadIdx.z+LS_APRON_SIZE;
    unsigned int windowIndex = (windowZIndex*blockDim.x) + threadIdx.x;

    ///////////////////////////////////////////
    // Load the lattice into shared memory. //
    ///////////////////////////////////////////

    // Shared memory to store the lattice segment. Each lattice site has four particles, eight bits for each particle.
    __shared__ unsigned int window[LS_Z_WINDOW_SIZE*LS_WORDS_PER_SITE];

    // Copy the x window from device memory into shared memory.
    copyZWindowFromLattice(inLattice, window, latticeIndex, latticeZIndex, latticeZSize, latticeXYSize, latticeXYZSize, windowIndex, windowZIndex);

    // Copy the z window from shared memory to device memory.
    copyZWindowToLattice(outLattice, window, latticeIndex, latticeXYZSize, windowIndex);
}

void cu_CopyZWindowAprons(unsigned int * host_inLattice, unsigned int * host_outLattice, unsigned int latticeXSize, unsigned int latticeYSize, unsigned int latticeZSize)
{
    void* inLattice;
    void* outLattice;
    CUDA_EXCEPTION_CHECK(hipMalloc(&inLattice, latticeXSize*latticeYSize*latticeZSize*LS_WORDS_PER_SITE*sizeof(unsigned int)));
    CUDA_EXCEPTION_CHECK(hipMalloc(&outLattice, latticeXSize*latticeYSize*latticeZSize*LS_WORDS_PER_SITE*sizeof(unsigned int)));
    CUDA_EXCEPTION_CHECK(hipMemcpy(inLattice, host_inLattice, latticeXSize*latticeYSize*latticeZSize*LS_WORDS_PER_SITE*sizeof(unsigned int), hipMemcpyHostToDevice));
    CUDA_EXCEPTION_CHECK(hipMemset(outLattice, 0xFF, latticeXSize*latticeYSize*latticeZSize*LS_WORDS_PER_SITE*sizeof(unsigned int)));

    unsigned int gridXSize = latticeXSize/LS_Z_BLOCK_X_SIZE;
    unsigned int gridYSize = latticeYSize;
    unsigned int gridZSize = latticeZSize/LS_Z_BLOCK_Z_SIZE;
    dim3 gridSize(gridXSize*gridYSize, gridZSize);
    dim3 threadBlockSize(LS_Z_BLOCK_X_SIZE, 1, LS_Z_BLOCK_Z_SIZE);
    CUDA_EXCEPTION_EXECUTE((cu_CopyZWindowAprons_kernel<<<gridSize,threadBlockSize>>>((unsigned int*)inLattice, (unsigned int*)outLattice, gridXSize, latticeXSize, latticeYSize, latticeZSize)));

    CUDA_EXCEPTION_CHECK(hipStreamSynchronize(0));
    CUDA_EXCEPTION_CHECK(hipMemcpy(host_outLattice, outLattice, latticeXSize*latticeYSize*latticeZSize*LS_WORDS_PER_SITE*sizeof(unsigned int), hipMemcpyDeviceToHost));

    CUDA_EXCEPTION_CHECK(hipFree(outLattice));
    CUDA_EXCEPTION_CHECK(hipFree(inLattice));
}

__global__ void cu_CopyZWindowAprons_kernel(const unsigned int* inLattice, unsigned int* outLattice, const unsigned int gridXSize, const unsigned int latticeXSize, const unsigned int latticeYSize, const unsigned int latticeZSize)
{
    unsigned int latticeXYSize = latticeXSize*latticeYSize;
    unsigned int latticeXYZSize = latticeXSize*latticeYSize*latticeZSize;

    __shared__ unsigned int bx, by, bz;
    calculateBlockPosition(&bx, &by, &bz, gridXSize);

    // Figure out the offset of this thread in the lattice and the lattice segment.
    unsigned int latticeZIndex = (bz*blockDim.z) + threadIdx.z;
    unsigned int latticeIndex = (latticeZIndex*latticeXYSize) + (by*latticeXSize) + (bx*blockDim.x) + threadIdx.x;
    unsigned int windowZIndex = threadIdx.z+LS_APRON_SIZE;
    unsigned int windowIndex = (windowZIndex*blockDim.x) + threadIdx.x;

    ///////////////////////////////////////////
    // Load the lattice into shared memory. //
    ///////////////////////////////////////////

    // Shared memory to store the lattice segment. Each lattice site has four particles, eight bits for each particle.
    __shared__ unsigned int window[LS_Z_WINDOW_SIZE*LS_WORDS_PER_SITE];

    // Copy the x window from device memory into shared memory.
    copyZWindowFromLattice(inLattice, window, latticeIndex, latticeZIndex, latticeZSize, latticeXYSize, latticeXYZSize, windowIndex, windowZIndex);

    __syncthreads();

    outLattice[latticeIndex] = 0;
    outLattice[latticeIndex+latticeXYZSize] = 0;

    // If this is the first part of the block, load the leading apron.
    if (windowZIndex < 2*LS_APRON_SIZE)
    {
        outLattice[latticeIndex] = window[windowIndex-(LS_Z_BLOCK_X_SIZE*LS_APRON_SIZE)];
        outLattice[latticeIndex+latticeXYZSize] = window[windowIndex-(LS_Z_BLOCK_X_SIZE*LS_APRON_SIZE)+LS_Z_WINDOW_SIZE];
    }

    // If this is the last part of the block, load the trailing apron.
    if (windowZIndex >= LS_Z_BLOCK_Z_SIZE)
    {
        outLattice[latticeIndex] = window[windowIndex+(LS_Z_BLOCK_X_SIZE*LS_APRON_SIZE)];
        outLattice[latticeIndex+latticeXYZSize] = window[windowIndex+(LS_Z_BLOCK_X_SIZE*LS_APRON_SIZE)+LS_Z_WINDOW_SIZE];
    }
}
